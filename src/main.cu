#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include "png_writer.cuh"

using namespace std;

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line
                  << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ void render(float *o_frame_buffer, int width, int height) {
    int pixel_x = threadIdx.x + blockIdx.x * blockDim.x;
    int pixel_y = threadIdx.y + blockIdx.y * blockDim.y;
    if ((pixel_x >= width) || (pixel_y >= height)) {
        return;
    }
    int pixel_index = pixel_y * width * 3 + pixel_x * 3;
    o_frame_buffer[pixel_index + 0] = float(pixel_x) / width;
    o_frame_buffer[pixel_index + 1] = float(pixel_y) / height;
    o_frame_buffer[pixel_index + 2] = 0.2;
}

void writer_to_file(const string &file_name, int width, int height, const float *float_buffer) {
    PngWriter png(width, height);

    float scalar = 256 - 0.0001;

    // set some pixels....
    for (int i = 0; i < width; ++i) {
        for (int k = 0; k < height; ++k) {
            size_t pixel_index = k * 3 * width + i * 3;

            int red = int(scalar * float_buffer[pixel_index + 0]);
            int green = int(scalar * float_buffer[pixel_index + 1]);
            int blue = int(scalar * float_buffer[pixel_index + 2]);

            png.set(i, k, red, green, blue); // set function assumes (0,0) is bottom left
        }
    }

    png.write(file_name);
}

int main() {
    int width = 1960;
    int height = 1080;
    int thread_width = 8;
    int thread_height = 8;

    std::cerr << "Rendering a " << width << "x" << height << " image ";
    std::cerr << "in " << thread_width << "x" << thread_height << " blocks.\n";

    // allocate FB
    float *frame_buffer;
    checkCudaErrors(hipMallocManaged((void **)&frame_buffer, 3 * sizeof(float) * width * height));

    clock_t start = clock();
    // Render our buffer
    dim3 blocks(width / thread_width + 1, height / thread_height + 1);
    dim3 threads(thread_width, thread_height);
    render<<<blocks, threads>>>(frame_buffer, width, height);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    double timer_seconds = ((double)(clock() - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    string file_name = "output.png";

    writer_to_file(file_name, width, height, frame_buffer);

    checkCudaErrors(hipFree(frame_buffer));

    cout << "image saved to `" << file_name << "`\n";

    return 0;
}
