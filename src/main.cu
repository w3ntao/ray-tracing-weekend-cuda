#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include "base/image.cuh"
#include "base/vector3.cuh"
#include "base/point3.cuh"
#include "base/ray.cuh"

using namespace std;

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)

void check_cuda(hipError_t result, char const *const func, const char *const file,
                int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at "
                  << file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ bool hit_sphere(const Point3 &center, float radius, const Ray &r) {
    Vector3 oc = r.o - center;
    float a = dot(r.d, r.d);
    float b = 2.0f * dot(oc, r.d);
    float c = dot(oc, oc) - radius * radius;
    float discriminant = b * b - 4.0f * a * c;
    return (discriminant > 0.0f);
}

__device__ Color color(const Ray &r) {
    if (hit_sphere(Point3(0, 0, -1), 0.5, r)) {
        return Color(1, 0, 0);
    }

    Vector3 unit_direction = r.d.normalize();
    float t = 0.5f * (unit_direction.y + 1.0f);
    auto result = (1.0f - t) * Vector3(1.0, 1.0, 1.0) + t * Vector3(0.5, 0.7, 1.0);

    return Color(result.x, result.y, result.z);
}

__global__ void render(Color *frame_buffer, int width, int height,
                       Point3 lower_left_corner, Vector3 horizontal, Vector3 vertical,
                       Point3 origin) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if ((x >= width) || (y >= height)) {
        return;
    }

    float u = float(x) / float(width);
    float v = float(y) / float(height);

    Ray ray(origin, (lower_left_corner + u * horizontal + v * vertical).to_vector());
    frame_buffer[y * width + x] = color(ray);
}

void writer_to_file(const string &file_name, int width, int height,
                    const Color *frame_buffer) {
    Image image(frame_buffer, width, height);
    image.writePNG(file_name);
}

int main() {
    int width = 1600;
    int height = 800;
    int thread_width = 8;
    int thread_height = 8;
    std::cerr << "Rendering a " << width << "x" << height << " image ";
    std::cerr << "in " << thread_width << "x" << thread_height << " blocks.\n";

    // allocate FB
    Color *frame_buffer;
    checkCudaErrors(
        hipMallocManaged((void **)&frame_buffer, sizeof(Color) * width * height));

    clock_t start = clock();
    // Render our buffer
    dim3 blocks(width / thread_width + 1, height / thread_height + 1, 1);
    dim3 threads(thread_width, thread_height, 1);

    render<<<blocks, threads>>>(frame_buffer, width, height, Point3(-2.0, -1.0, -1.0),
                                Vector3(4.0, 0.0, 0.0), Vector3(0.0, 2.0, 0.0),
                                Point3(0.0, 0.0, 0.0));

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    double timer_seconds = ((double)(clock() - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    string file_name = "output.png";

    writer_to_file(file_name, width, height, frame_buffer);

    checkCudaErrors(hipFree(frame_buffer));

    cout << "image saved to `" << file_name << "`\n";

    return 0;
}
