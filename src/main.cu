#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <cfloat>
#include <hiprand/hiprand_kernel.h>
#include "base/image.h"
#include "base/sphere.h"
#include "base/world.h"
#include "base/camera.h"

using namespace std;

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)

void check_cuda(hipError_t result, char const *const func, const char *const file,
                int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":"
                  << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ void create_world(Shape **d_list, World **d_world, Camera **d_camera) {
    *(d_list + 0) = new Sphere(Point3(0, 0, -1), 0.5);
    *(d_list + 1) = new Sphere(Point3(0, -100.5, -1), 100);
    *d_world = new World(d_list, 2);
    *d_camera = new Camera();
}

__global__ void free_world(World **d_world, Camera **d_camera) {
    for (int idx = 0; idx < (*d_world)->size; idx++) {
        delete (*d_world)->list[idx];
    }
    delete *d_world;
    delete *d_camera;
}

__device__ Vector3 random_in_unit_sphere(hiprandState *local_rand_state) {
    Vector3 p;
    do {
        auto random_vector =
            Vector3(hiprand_uniform(local_rand_state), hiprand_uniform(local_rand_state),
                    hiprand_uniform(local_rand_state));
        p = 2.0f * random_vector - Vector3(1, 1, 1);
    } while (p.squared_length() >= 1.0f);
    return p;
}

__device__ Color color(const Ray &r, World **world, hiprandState *local_rand_state) {
    Ray cur_ray = r;
    float cur_attenuation = 1.0f;
    for (int i = 0; i < 50; i++) {
        Intersection intersection;
        if ((*world)->intersect(intersection, cur_ray, 0.001f, FLT_MAX)) {
            Point3 target =
                intersection.p + intersection.n + random_in_unit_sphere(local_rand_state);
            cur_attenuation *= 0.5f;
            cur_ray = Ray(intersection.p, target - intersection.p);
            continue;
        }

        Vector3 unit_direction = cur_ray.d.normalize();
        float t = 0.5f * (unit_direction.y + 1.0f);
        Vector3 c = (1.0f - t) * Vector3(1.0, 1.0, 1.0) + t * Vector3(0.5, 0.7, 1.0);

        auto result = cur_attenuation * c;
        return Color(result.x, result.y, result.z);
    }
    return Color(0.0, 0.0, 0.0); // exceeded recursion
}

__global__ void render_init(int width, int height, hiprandState *rand_state) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if ((x >= width) || (y >= height))
        return;
    int pixel_index = y * width + x;
    // Each thread gets same seed, a different sequence number, no offset
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render(Color *frame_buffer, int width, int height, int num_samples, Camera **camera,
                       World **world, hiprandState *rand_state) {
    uint x = threadIdx.x + blockIdx.x * blockDim.x;
    uint y = threadIdx.y + blockIdx.y * blockDim.y;
    if ((x >= width) || (y >= height)) {
        return;
    }
    int pixel_index = y * width + x;
    hiprandState local_rand_state = rand_state[pixel_index];

    Color final_color(0, 0, 0);
    for (int s = 0; s < num_samples; s++) {
        float u = float(x + hiprand_uniform(&local_rand_state)) / float(width);
        float v = float(y + hiprand_uniform(&local_rand_state)) / float(height);
        final_color += color((*camera)->get_ray(u, v), world, &local_rand_state);
    }

    rand_state[pixel_index] = local_rand_state;
    final_color /= float(num_samples);

    final_color = Color(sqrt(final_color.r), sqrt(final_color.g), sqrt(final_color.b));
    frame_buffer[y * width + x] = final_color;
}

void writer_to_file(const string &file_name, int width, int height, const Color *frame_buffer) {
    Image image(frame_buffer, width, height);
    image.flip();
    image.writePNG(file_name);
}

int main() {
    int width = 1600;
    int height = 800;
    int thread_width = 8;
    int thread_height = 8;
    int num_samples = 100;

    std::cerr << "Rendering a " << width << "x" << height
              << " image (samples per pixel: " << num_samples << ") ";
    std::cerr << "in " << thread_width << "x" << thread_height << " blocks.\n";

    // allocate random state
    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, width * height * sizeof(hiprandState)));

    // allocate FB
    Color *frame_buffer;
    checkCudaErrors(hipMallocManaged((void **)&frame_buffer, sizeof(Color) * width * height));

    Shape **d_list;
    checkCudaErrors(hipMalloc((void **)&d_list, 2 * sizeof(Shape *)));
    World **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(World *)));
    Camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(Camera *)));
    create_world<<<1, 1>>>(d_list, d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start = clock();
    // Render our buffer
    dim3 blocks(width / thread_width + 1, height / thread_height + 1, 1);
    dim3 threads(thread_width, thread_height, 1);

    render_init<<<blocks, threads>>>(width, height, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    render<<<blocks, threads>>>(frame_buffer, width, height, num_samples, d_camera, d_world,
                                d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    double timer_seconds = ((double)(clock() - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    string file_name = "output.png";
    writer_to_file(file_name, width, height, frame_buffer);

    free_world<<<1, 1>>>(d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(frame_buffer));

    cout << "image saved to `" << file_name << "`\n";

    return 0;
}
