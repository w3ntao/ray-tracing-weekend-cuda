#include "hip/hip_runtime.h"
#include <time.h>
#include <iostream>
#include <fstream>
#include <string>

#include <png_writer.h>

using namespace std;

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line
                  << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ void render(float *fb, int max_x, int max_y) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y))
        return;
    int pixel_index = j * max_x * 3 + i * 3;
    fb[pixel_index + 0] = float(i) / max_x;
    fb[pixel_index + 1] = float(j) / max_y;
    fb[pixel_index + 2] = 0.2;
}

void writer_to_file(const string &file_name, int nx, int ny, const float *fb) {
    PngWriter png(nx, ny);

    // set some pixels....
    for (int i = 0; i < nx; ++i) {
        for (int j = 0; j < ny; ++j) {

            size_t pixel_index = j * 3 * nx + i * 3;
            float r = fb[pixel_index + 0];
            float g = fb[pixel_index + 1];
            float b = fb[pixel_index + 2];
            int ir = int(255.99 * r);
            int ig = int(255.99 * g);
            int ib = int(255.99 * b);

            png.set(i, j, ir, ig, ib); // set function assumes (0,0) is bottom left
        }
    }

    png.write(file_name);
}

int main() {
    int nx = 1960;
    int ny = 1080;
    int tx = 16;
    int ty = 16;

    std::cerr << "Rendering a " << nx << "x" << ny << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx * ny;
    size_t fb_size = 3 * num_pixels * sizeof(float);

    // allocate FB
    float *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    clock_t start = clock();
    // Render our buffer
    dim3 blocks(nx / tx + 1, ny / ty + 1);
    dim3 threads(tx, ty);
    render<<<blocks, threads>>>(fb, nx, ny);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    double timer_seconds = ((double)(clock() - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    string file_name = "output.png";

    writer_to_file(file_name, nx, ny, fb);

    checkCudaErrors(hipFree(fb));

    cout << "image saved to `" << file_name << "`\n";

    return 0;
}
